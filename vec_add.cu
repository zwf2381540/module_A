#include <stdio.h>
#include "hip/hip_runtime.h"

// Device code
// 将数组 A 与 B 中的元素相加存入数组 C 中, N 为数组中元素的数量
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
int main()
{
    int N = 1024;
    size_t size = N * sizeof(float);

    // 定义 host 端数组 h_A, h_B, h_C
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    // ...

    // 定义 device 端数组 d_A, d_B, d_C
    // 并在 GPU 中为它们分配对应的显存空间
    // &d_A 的类型为 float **, 此处将其强制转换为 void** 类型.
    float* d_A;
    hipMalloc((void **)&d_A, size);
    float* d_B;
    hipMalloc((void **)&d_B, size);
    float* d_C;
    hipMalloc((void **)&d_C, size);

    // 利用 cudaMemcpy 函数将 host 端 A,B 的值复制到对应的 GPU 内存中
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    // 启动定义好的 GPU kernel，实现数组相加
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // cudaDeviceSynchronize() 会阻塞当前程序的执行, 直到所有线程都执行完 kernel
    // 避免后面的 CPU 代码在 kernel 结束前就执行.
    hipDeviceSynchronize();

    // 调用 cudaMemcpy 函数将 GPU 端计算结果复制到 CPU 端
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 打印数组相加结果
    for(int i=0; i<N; ++i){
        printf("%d ", h_C[i]);
    }
    printf("\n");

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // reset device before you leave
    hipDeviceReset();
    return 0;
}
